#include "hip/hip_runtime.h"
#include "kawpow.h"
#include "logging.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <mutex>
#include <cstdio>
#include <chrono>
#include <iomanip>
#include <sstream>
#include <unordered_map>
#include <mutex>
extern "C" {
    #include "libethash/ethash.h"
}

// Define constants once at the top
#define PROGPOW_LANES 16
#define PROGPOW_REGS 32
#define PROGPOW_LOOP_COUNT 64
#define PROGPOW_DAG_ITEM_SIZE 64
#define PROGPOW_DAG_PARENTS 256

// --- FNV1a Hashing Helper ---
#define FNV_PRIME 0x01000193


__constant__ uint32_t d_ravencoin_kawpow[15];
// const uint64_t fixed_difficulty = 256; // tweak as needed


__device__ inline uint32_t fnv1a_32(uint32_t a, uint32_t b) { return (a ^ b) * FNV_PRIME; }
uint32_t fnv1a_32_cpu(uint32_t a, uint32_t b) { return (a ^ b) * FNV_PRIME; }

// --- Keccak Hashing & Helper Implementation for GPU ---
__device__ inline uint64_t rotate_left(uint64_t x, uint8_t n) { return (x << n) | (x >> (64 - n)); }
__device__ inline uint32_t rotate_right(uint32_t x, uint32_t n) { return (x >> n) | (x << (32 - n)); }

// --- FIXED: Proper endianness handling ---
__device__ inline uint32_t byteswap_32(uint32_t x) { 
    return ((x << 24) & 0xff000000) | 
           ((x << 8)  & 0x00ff0000) | 
           ((x >> 8)  & 0x0000ff00) | 
           ((x >> 24) & 0x000000ff); 
}

__device__ inline uint64_t byteswap_64(uint64_t x) {
    return ((x << 56) & 0xff00000000000000ULL) |
           ((x << 40) & 0x00ff000000000000ULL) |
           ((x << 24) & 0x0000ff0000000000ULL) |
           ((x << 8)  & 0x000000ff00000000ULL) |
           ((x >> 8)  & 0x00000000ff000000ULL) |
           ((x >> 24) & 0x0000000000ff0000ULL) |
           ((x >> 40) & 0x000000000000ff00ULL) |
           ((x >> 56) & 0x00000000000000ffULL);
}

__device__ static void keccak_f1600(uint64_t* state) {
    const uint64_t keccak_round_constants[24] = {
        0x0000000000000001, 0x0000000000008082, 0x800000000000808a, 0x8000000080008000,
        0x000000000000808b, 0x0000000080000001, 0x8000000080008081, 0x8000000000008009,
        0x000000000000008a, 0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
        0x000000008000808b, 0x800000000000008b, 0x8000000000008089, 0x8000000000008003,
        0x8000000000008002, 0x8000000000000080, 0x000000000000800a, 0x800000008000000a,
        0x8000000080008081, 0x8000000000008080, 0x0000000080000001, 0x8000000080008008
    };
    
    for (int i = 0; i < 24; ++i) {
        uint64_t C[5], D;
        for (int x = 0; x < 5; ++x) C[x] = state[x] ^ state[x + 5] ^ state[x + 10] ^ state[x + 15] ^ state[x + 20];
        for (int x = 0; x < 5; ++x) {
            D = C[(x + 4) % 5] ^ rotate_left(C[(x + 1) % 5], 1);
            for (int y = 0; y < 25; y += 5) state[y + x] ^= D;
        }
        uint64_t current = state[1];
        for (int x = 0; x < 24; ++x) {
            int r = ((x + 1) * (x + 2) / 2) % 64;
            int lane = (2 * (x % 5) + 3 * ((x / 5) % 5)) % 5;
            uint64_t temp = state[1 + lane];
            state[1 + lane] = rotate_left(current, r);
            current = temp;
        }
        // Chi step
        for (int y = 0; y < 25; y += 5) {
            uint64_t temp[5];
            for (int x = 0; x < 5; ++x) temp[x] = state[y + x];
            for (int x = 0; x < 5; ++x) state[y + x] = temp[x] ^ ((~temp[(x + 1) % 5]) & temp[(x + 2) % 5]);
        }
        state[0] ^= keccak_round_constants[i];
    }
}

// Global DAG cache

struct DagCache {
    void* d_dag = nullptr;
    size_t size = 0;
    uint64_t epoch = UINT64_MAX;
    std::mutex mutex;

    // prevent copying
    DagCache(const DagCache&) = delete;
    DagCache& operator=(const DagCache&) = delete;

    // allow moving
    DagCache(DagCache&&) = default;
    DagCache& operator=(DagCache&&) = default;

    // default constructor
    DagCache() = default;
};


static std::unordered_map<int, DagCache> g_dag_caches;
static std::mutex g_dag_mutex;

// ===================================================================================
// == DAG & Light Cache Generation
// ===================================================================================
uint64_t get_kawpow_dag_size(uint64_t epoch) {
    const uint64_t dag_size_initial = 1ULL * 1024 * 1024 * 1024;
    const uint64_t dag_size_growth = 8ULL * 1024 * 1024;
    return dag_size_initial + (epoch * dag_size_growth);
}

void* generate_kawpow_light_cache(uint64_t epoch, const char* seed_hash_hex, size_t& cache_size) {
    const uint32_t light_cache_initial_size = 16 * 1024 * 1024;
    const uint32_t light_cache_growth = 256 * 1024;
    const uint32_t words_per_item = 16;
    cache_size = light_cache_initial_size + (epoch * light_cache_growth);
    uint32_t num_cache_items = cache_size / sizeof(uint32_t);
    uint32_t* h_cache = (uint32_t*)malloc(cache_size);
    if (!h_cache) { LOG_ERROR << "Failed to allocate memory for light cache on CPU."; return nullptr; }
    
    // Parse seed hash correctly
    uint32_t seed[8];
    for (int i = 0; i < 8; ++i) {
        sscanf(seed_hash_hex + (i * 8), "%8x", &seed[i]);
    }
    
    // Initialize cache with seed
    for (int i = 0; i < 8; ++i) h_cache[i] = seed[i];
    for (uint32_t i = 8; i < words_per_item; ++i) h_cache[i] = fnv1a_32_cpu(h_cache[i - 8], h_cache[i - 7]);
    
    // Generate remaining cache items
    for (uint32_t i = words_per_item; i < num_cache_items; ++i) {
        h_cache[i] = fnv1a_32_cpu(h_cache[i - words_per_item], h_cache[i - (words_per_item - 1)]);
    }
    
    LOG_INFO << "Successfully generated " << cache_size / (1024*1024) << " MB light cache for epoch " << epoch;
    return h_cache;
}

// ===================================================================================
// == CORRECTED DAG Generation Kernel (Ethash/KawPoW Standard)
// ===================================================================================
__global__ void generate_dag_kernel(uint32_t* d_dag, const uint32_t* d_cache, uint32_t num_dag_items, uint32_t num_cache_items)
{
    const uint32_t node_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (node_index >= num_dag_items) {
        return;
    }

    // Constants for standard Ethash DAG generation
    const uint32_t HASH_WORDS = 16; // 512 bits / 32 bits per word
    const uint32_t DAG_PARENTS = 256;

    // The number of 64-byte items in the light cache
    const uint32_t num_cache_words = num_cache_items / HASH_WORDS;

    // Calculate the starting cache index for this DAG node
    const uint32_t start_node_index = node_index % num_cache_words;
    const uint32_t* cache_node_ptr = d_cache + start_node_index * HASH_WORDS;

    // Initialize the mix state from the light cache
    uint32_t mix[HASH_WORDS];
    for (uint32_t i = 0; i < HASH_WORDS; ++i) {
        mix[i] = cache_node_ptr[i];
    }

    // Main DAG generation loop
    for (uint32_t i = 0; i < DAG_PARENTS; ++i) {
        // Calculate the index of the parent node to fetch from the light cache
        // This uses standard modular arithmetic, NOT the complex "laned" version.
        uint32_t parent_index = fnv1a_32(node_index ^ i, mix[i % HASH_WORDS]) % num_cache_words;
        const uint32_t* parent_ptr = d_cache + parent_index * HASH_WORDS;

        // Mix the parent data into the current state
        for (uint32_t j = 0; j < HASH_WORDS; ++j) {
            mix[j] = fnv1a_32(mix[j], parent_ptr[j]);
        }
    }

    // Final step: In ProgPoW/KawPoW, the DAG item is the mix itself.
    // (In standard Ethash, there would be a final hash here, but not for KawPoW's DAG).
    // Copy the final mix to the global DAG memory.
    uint32_t* dag_item_ptr = d_dag + node_index * HASH_WORDS;
    for (uint32_t i = 0; i < HASH_WORDS; ++i) {
        dag_item_ptr[i] = mix[i];
    }
}

// ===================================================================================
// == Final Hashing Kernel and Helpers
// ===================================================================================
struct kiss99_rng {
    uint32_t z, w, jsr, jcong;
    __device__ void seed(uint32_t s) { z = w = jsr = jcong = s; }
    __device__ uint32_t get() {
        z = 36969 * (z & 65535) + (z >> 16); 
        w = 18000 * (w & 65535) + (w >> 16); 
        uint32_t mwc = (z << 16) + w;
        jsr ^= (jsr << 17); jsr ^= (jsr >> 13); jsr ^= (jsr << 5); 
        jcong = 69069 * jcong + 1234567;
        return (mwc ^ jcong) + jsr;
    }
};

struct uint256_t { uint32_t val[8]; };

__device__ inline bool is_less_or_equal(const uint256_t& a, const uint256_t& b) {
    for (int i = 7; i >= 0; --i) { 
        if (a.val[i] < b.val[i]) return true; 
        if (a.val[i] > b.val[i]) return false; 
    } 
    return true;
}

// In src/kawpow.cu, replace your entire old kernel with this one.

__global__ void kawpow_kernel(
    uint64_t* d_result_nonce, char* d_result_mix_hash,
    const char* d_header_hash, uint64_t start_nonce,
    const uint32_t* d_dag, const char* d_target_hex)
{
    uint64_t nonce = start_nonce + blockIdx.x * blockDim.x + threadIdx.x;

    if (*d_result_nonce != 0) {
        return;
    }

    // --- Step 1: Initial Keccak Hash ---
    uint64_t keccak_state[25] = {0};
    uint32_t* keccak_state_32 = (uint32_t*)keccak_state;

    // Load header hash (32 bytes)
    for (int i = 0; i < 8; ++i) {
        keccak_state_32[i] = ((uint32_t*)d_header_hash)[i];
    }
    // Load nonce (8 bytes)
    *(uint64_t*)(&keccak_state_32[8]) = nonce;
    // Load "RAVENCOINKAWPOW" constant (60 bytes)
    for (int i = 0; i < 15; ++i) {
        keccak_state_32[10 + i] = d_ravencoin_kawpow[i];
    }

    // Keccak-f800 is used, which means 100 bytes of input.
    // We use Keccak-f1600, so we must set the correct padding for a 100-byte message.
    // 100 bytes = 12.5 uint64_t words.
    // The padding starts at byte 100, which is keccak_state[12].
    keccak_state[12] |= 0x0000000000000001; // Padding: 1
    keccak_state[16] |= 0x8000000000000000; // Padding: 1...0...1

    keccak_f1600(keccak_state);

    // --- Step 2: Main ProgPoW Loop (Your existing code is likely okay here for now) ---
    uint32_t mix[PROGPOW_REGS];
    for (int i = 0; i < 16; ++i) mix[i] = ((uint32_t*)keccak_state)[i];
    for (int i = 16; i < PROGPOW_REGS; ++i) mix[i] = mix[i - 16] * FNV_PRIME;

    kiss99_rng rng;
    rng.seed(mix[0] ^ mix[1]);
    uint32_t dag_lookup_state[PROGPOW_LANES];
    for (int i = 0; i < PROGPOW_LANES; ++i) dag_lookup_state[i] = mix[i];

    for (uint32_t i = 0; i < PROGPOW_LOOP_COUNT; ++i) {
        uint32_t dag_item_index = rng.get() % (PROGPOW_LANES * PROGPOW_REGS);
        for (uint32_t l = 0; l < PROGPOW_LANES; ++l) {
            uint32_t dag_word_index = dag_item_index * 16 + l;
            dag_lookup_state[l] = fnv1a_32(dag_lookup_state[l], d_dag[dag_word_index]);
        }
        uint32_t src_rand = rng.get(), dst_rand = rng.get(), sel_rand = rng.get();
        uint32_t src1 = src_rand % PROGPOW_REGS, src2 = (src1 + (src_rand >> 16) % (PROGPOW_REGS - 1) + 1) % PROGPOW_REGS;
        uint32_t dst = dst_rand % PROGPOW_REGS, sel = sel_rand % 4;
        switch (sel) {
            case 0: mix[dst] *= mix[src1]; break;
            case 1: mix[dst] += mix[src2]; break;
            case 2: mix[dst] = rotate_right(mix[src1], mix[src2] & 31); break;
            case 3: mix[dst] ^= mix[src2]; break;
        }
    }

    // --- Step 3: Final Mix Hash Aggregation (Corrected based on XMRig) ---
    const uint32_t fnv_offset_basis = 0x811c9dc5;
    uint32_t final_mix_hash[8];
    for (int i = 0; i < 8; ++i) {
        final_mix_hash[i] = fnv_offset_basis;
    }
    // Each thread calculates its lane hash and combines it into the final hash
    // This assumes PROGPOW_LANES is 16
    for (int i = 0; i < PROGPOW_REGS; i += 2) {
        uint32_t h1 = fnv1a_32(fnv_offset_basis, mix[i]);
        uint32_t h2 = fnv1a_32(h1, mix[i+1]);
        final_mix_hash[i % 8] = fnv1a_32(final_mix_hash[i % 8], h2);
    }


    // --- Step 4: Final Keccak Hash ---
    // Reset state and prepare for final hash
    for(int i = 0; i < 25; ++i) keccak_state[i] = 0;

    // Load header hash (32 bytes)
    for (int i = 0; i < 8; ++i) {
        keccak_state_32[i] = ((uint32_t*)d_header_hash)[i];
    }
    // Load the final mix hash we just calculated (32 bytes)
    for (int i = 0; i < 8; ++i) {
        keccak_state_32[8 + i] = final_mix_hash[i];
    }
    // Load a different part of the "RAVENCOINKAWPOW" constant (36 bytes)
    for (int i = 0; i < 9; ++i) {
        keccak_state_32[16 + i] = d_ravencoin_kawpow[i];
    }

    // Padding for a 100-byte message (32+32+36)
    keccak_state[12] |= 0x0000000000000001;
    keccak_state[16] |= 0x8000000000000000;

    keccak_f1600(keccak_state);

    // --- Step 5: Comparison (Using your existing endian-correct logic) ---
    uint256_t target_val, hash_val;
    for (int i = 0; i < 8; ++i) {
        uint32_t val = 0;
        for (int j = 0; j < 8; ++j) {
            char c = d_target_hex[i * 8 + j];
            uint32_t nibble = (c >= 'a') ? (c - 'a' + 10) : (c - '0');
            val = (val << 4) | nibble;
        }
        target_val.val[7 - i] = val;
    }
    for (int i = 0; i < 8; ++i) {
        hash_val.val[7 - i] = byteswap_32(keccak_state_32[i]);
    }

    if (is_less_or_equal(hash_val, target_val)) {
        if (atomicCAS((unsigned long long*)d_result_nonce, 0, (unsigned long long)nonce) == 0) {
            for (int i = 0; i < 32; ++i) {
                d_result_mix_hash[i] = ((char*)final_mix_hash)[i];
            }
        }
    }
}

// ===================================================================================
// == DAG Management and Main Search Function
// ===================================================================================
void* get_dag(uint64_t block_number, const char* seed_hash_hex, uint64_t& dag_size, int device_id) {
    uint64_t epoch = block_number / 7500;

    // 🔐 Step 1: Ensure a DagCache exists for this device
    {
        std::lock_guard<std::mutex> lock(g_dag_mutex);
        g_dag_caches.try_emplace(device_id);

    }

    // 🔐 Step 2: Work with this device’s DAG cache
    DagCache& cache = g_dag_caches[device_id];
    std::lock_guard<std::mutex> lock(cache.mutex);

    if (cache.epoch == epoch && cache.d_dag != nullptr) {
        dag_size = cache.size;
        return cache.d_dag;
    }

    LOG_INFO << "Generating new DAG for epoch " << epoch << " (Block: " << block_number << ")";

    size_t cache_size;
    void* h_cache = generate_kawpow_light_cache(epoch, seed_hash_hex, cache_size);
    if (!h_cache) return nullptr;

    dag_size = get_kawpow_dag_size(epoch);
    LOG_INFO << "Calculated DAG size: " << dag_size / (1024 * 1024) << " MB";

    // Clean up old DAG
    if (cache.d_dag != nullptr) {
        hipFree(cache.d_dag);
    }

    uint32_t* d_cache = nullptr;
    uint32_t* d_dag = nullptr;

    if (hipMalloc(&d_dag, dag_size) != hipSuccess) {
        LOG_ERROR << "GPU DAG Malloc failed";
        free(h_cache);
        return nullptr;
    }

    if (hipMalloc(&d_cache, cache_size) != hipSuccess) {
        LOG_ERROR << "GPU Cache Malloc failed";
        free(h_cache);
        hipFree(d_dag);
        return nullptr;
    }

    hipMemcpy(d_cache, h_cache, cache_size, hipMemcpyHostToDevice);
    free(h_cache);

    dim3 threads_per_block(256);
    dim3 num_blocks((dag_size / 64 + 255) / 256);
    generate_dag_kernel<<<num_blocks, threads_per_block>>>(d_dag, d_cache, dag_size / 64, cache_size / sizeof(uint32_t));
    hipDeviceSynchronize();
    hipFree(d_cache);

    cache.epoch = epoch;
    cache.size = dag_size;
    cache.d_dag = d_dag;

    return d_dag;
}

struct uint256 {
    uint32_t val[8]; // little endian or big endian - be consistent
    
    // Divide by uint64_t small number, returns quotient
    uint256 operator/(uint64_t divisor) const {
        uint256 result = {};
        uint64_t remainder = 0;
        for (int i = 7; i >= 0; --i) {
            uint64_t part = (remainder << 32) | val[i];
            result.val[i] = static_cast<uint32_t>(part / divisor);
            remainder = part % divisor;
        }
        return result;
    }
};

uint256 parse_hex_to_uint256(const std::string& hex) {
    uint256 result = {};
    // parse 64 hex chars to 8 x 32-bit words
    for (int i = 0; i < 8; ++i) {
        sscanf(hex.c_str() + i * 8, "%8x", &result.val[7 - i]); // big endian parsing
    }
    return result;
}

std::string uint256_to_hex(const uint256& value) {
    std::stringstream ss;
    ss << std::hex << std::setfill('0');
    for (int i = 7; i >= 0; --i) {
        ss << std::setw(8) << value.val[i];
    }
    return ss.str();
}

extern "C" void kawpow_cuda_search(
    const char* header_hash, const char* seed_hash, uint64_t block_number, 
    uint64_t start_nonce, int device_id, int intensity, 
    KawPow* kawpow_instance, const char* target)
{
    hipSetDevice(device_id);
    
    uint64_t dag_size;
    uint32_t* d_dag = static_cast<uint32_t*>(get_dag(block_number, seed_hash, dag_size, device_id));
    if (!d_dag) { 
        LOG_ERROR << "Device " << device_id << ": Failed to get DAG."; 
        return; 
    }

    // Allocate GPU memory for kernel parameters
    char *d_header_hash, *d_target_hex, *d_result_mix_hash;
    uint64_t* d_result_nonce;
    
    hipMalloc(&d_header_hash, 32);
    hipMalloc(&d_target_hex, 64);
    hipMalloc(&d_result_nonce, sizeof(uint64_t));
    hipMalloc(&d_result_mix_hash, 32);
    
    // Copy data to GPU
    hipMemcpy(d_header_hash, header_hash, 32, hipMemcpyHostToDevice);
    hipMemcpy(d_target_hex, target, 64, hipMemcpyHostToDevice);
    hipMemset(d_result_nonce, 0, sizeof(uint64_t));

    LOG_INFO << "Device " << device_id << ": Starting search loop for block " << block_number;
    
    uint64_t current_nonce = start_nonce;
    dim3 threads_per_block(256);
    dim3 num_blocks(intensity);

    // 1. Parse pool max target string (hardcoded or from mining.set_target)
    // std::string pool_max_target = "00000000ffff0000000000000000000000000000000000000000000000000000";
    std::string pool_max_target = "00000001ffffffffffffffffffffffffffffffffffffffffffffffffffffffff";

    // 2. Convert to uint256 (or similar big int struct)
    uint256 max_target = parse_hex_to_uint256(pool_max_target);

    // 3. fixed difficulty
    uint64_t fixed_difficulty = 256;

    // 4. Calculate fixed share target
    uint256 fixed_target = max_target / fixed_difficulty;

    // 5. Format fixed_target back to hex string
    std::string fixed_target_hex = uint256_to_hex(fixed_target);

    // 6. Copy fixed_target_hex to GPU as d_target_hex instead of the param `target`
    hipMemcpy(d_target_hex, fixed_target_hex.c_str(), 64, hipMemcpyHostToDevice);
    auto start_time = std::chrono::high_resolution_clock::now();
    uint64_t total_hashes = 0;

    while (kawpow_instance->should_continue()) {
        kawpow_kernel<<<num_blocks, threads_per_block>>>(
            d_result_nonce, d_result_mix_hash, d_header_hash, current_nonce,
            d_dag, d_target_hex
        );

        uint64_t h_result_nonce = 0;
        hipMemcpy(&h_result_nonce, d_result_nonce, sizeof(uint64_t), hipMemcpyDeviceToHost);

        if (h_result_nonce != 0) {
            char h_mix_hash[32];
            hipMemcpy(h_mix_hash, d_result_mix_hash, 32, hipMemcpyDeviceToHost);
            
            // === FIXED: Proper nonce and mix hash formatting ===
            std::stringstream nonce_ss;
            nonce_ss << std::hex << std::setfill('0') << std::setw(16) << h_result_nonce;
            
            std::stringstream mix_ss;
            for(int i = 0; i < 32; ++i) {
                mix_ss << std::hex << std::setfill('0') << std::setw(2) << (int)(unsigned char)h_mix_hash[i];
            }
            
            LOG_INFO << "Device " << device_id << ": Found valid share! Nonce: " << nonce_ss.str();
            kawpow_instance->submit_share(nonce_ss.str(), mix_ss.str());
            
            // Reset for next search
            hipMemset(d_result_nonce, 0, sizeof(uint64_t));
        }
        
        current_nonce += (uint64_t)num_blocks.x * threads_per_block.x;
        total_hashes += (uint64_t)num_blocks.x * threads_per_block.x;

        auto now = std::chrono::high_resolution_clock::now();
        auto seconds_passed = std::chrono::duration_cast<std::chrono::seconds>(now - start_time).count();

        if (seconds_passed >= 5) {
            double hashrate = (double)total_hashes / seconds_passed;
            LOG_INFO << "Device " << device_id << ": ~" << (uint64_t)(hashrate / 1000000.0) << " MH/s";
            // Reset counters
            total_hashes = 0;
            start_time = now;
        }

    }

    // Cleanup
    hipFree(d_header_hash); 
    hipFree(d_target_hex); 
    hipFree(d_result_nonce); 
    hipFree(d_result_mix_hash);
    
    LOG_INFO << "Device " << device_id << ": Search loop finished.";
}

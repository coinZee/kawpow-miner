#include "hip/hip_runtime.h"
#include "kawpow.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <thread>
#include <chrono>
#include "libethash/ethash.h"
#include "libethash/ethash_internal.h"

// CUDA kernel - device code only, no logging here
__global__ void kawpow_kernel(const char* header_hash, uint64_t start_nonce, uint32_t* result, uint32_t* d_dag, uint64_t dag_size) {
    uint64_t nonce = start_nonce + blockIdx.x * blockDim.x + threadIdx.x;
    
    // In a real miner, the full KawPoW hashing logic would be implemented here.
    // This includes DAG access and multiple rounds of hashing.
    
    // Simplified simulation of DAG access to make sure we're using the DAG memory
    uint32_t idx = nonce % (dag_size / sizeof(uint32_t));
    uint32_t dag_item = d_dag[idx];
    
    // Simulate finding a valid nonce based on DAG value
    if ((dag_item + nonce) % 1000000 == 0) { // Simulate finding a valid nonce
        *result = nonce;
    }
}

// Simple logging function that doesn't use complex formatting
void cuda_log(const char* level, const char* color, const std::string& message) {
    std::cout << "\033[" << color << "m[" << level << "] " << message << "\033[0m" << std::endl;
}

// Function to log VRAM usage
void log_vram_usage(int device_id) {
    size_t free_mem, total_mem;
    hipMemGetInfo(&free_mem, &total_mem);
    size_t used_mem = total_mem - free_mem;
    cuda_log("CUDA", "36", "Device " + std::to_string(device_id) + " VRAM usage: " + std::to_string(used_mem / (1024 * 1024)) + " MB / " + std::to_string(total_mem / (1024 * 1024)) + " MB");
}

// Host code with logging
void kawpow_cuda_search(const char* header_hash, uint64_t start_nonce, int device_id, int intensity) {
    cuda_log("CUDA", "36", "Starting CUDA search on device " + std::to_string(device_id) + 
                           " with intensity " + std::to_string(intensity));
    
    hipError_t err = hipSetDevice(device_id);
    if (err != hipSuccess) {
        cuda_log("ERROR", "31", "Failed to set CUDA device " + std::to_string(device_id) + 
                               ": " + std::string(hipGetErrorString(err)));
        return;
    }

    // Get device properties
    hipDeviceProp_t deviceProp;
    err = hipGetDeviceProperties(&deviceProp, device_id);
    if (err != hipSuccess) {
        cuda_log("ERROR", "31", "Failed to get device properties: " + 
                               std::string(hipGetErrorString(err)));
        return;
    }
    
    std::string device_info = "Using device: ";
    device_info += deviceProp.name;
    device_info += " with compute capability ";
    device_info += std::to_string(deviceProp.major);
    device_info += ".";
    device_info += std::to_string(deviceProp.minor);
    cuda_log("CUDA", "36", device_info);

    // Generate and load DAG (simplified, using ethash for KawPoW DAG)
    cuda_log("CUDA", "36", "Generating and loading DAG...");
    
    // Use current block number to determine epoch
    uint64_t current_block = 3929879; // This should come from the stratum job
    uint64_t epoch = current_block / ETHASH_EPOCH_LENGTH;
    
    ethash_light_t light = ethash_light_new(epoch);
    if (!light) {
        cuda_log("ERROR", "31", "Failed to generate DAG");
        return;
    }
    
    // Calculate full DAG size
    uint64_t full_dag_size = ethash_get_datasize(current_block);
    
    // Allocate DAG on GPU
    uint32_t* d_dag;
    size_t dag_size = full_dag_size; // Use full DAG size, not just cache
    err = hipMalloc(&d_dag, dag_size);
    if (err != hipSuccess) {
        cuda_log("ERROR", "31", "Failed to allocate DAG memory: " + std::string(hipGetErrorString(err)));
        ethash_light_delete(light);
        return;
    }
    
    // Generate full DAG in host memory
    void* full_dag = malloc(full_dag_size);
    if (!full_dag) {
        cuda_log("ERROR", "31", "Failed to allocate host memory for full DAG");
        hipFree(d_dag);
        ethash_light_delete(light);
        return;
    }
    
    // Compute full DAG
    if (!ethash_compute_full_data(full_dag, full_dag_size, light, nullptr)) {
        cuda_log("ERROR", "31", "Failed to compute full DAG");
        free(full_dag);
        hipFree(d_dag);
        ethash_light_delete(light);
        return;
    }
    
    // Copy DAG to GPU
    err = hipMemcpy(d_dag, full_dag, dag_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        cuda_log("ERROR", "31", "Failed to copy DAG to device: " + std::string(hipGetErrorString(err)));
        free(full_dag);
        hipFree(d_dag);
        ethash_light_delete(light);
        return;
    }
    
    // Free host DAG memory
    free(full_dag);
    
    cuda_log("CUDA", "36", "DAG loaded successfully, size: " + std::to_string(dag_size / (1024 * 1024)) + " MB");

    // Start periodic VRAM logging thread
    std::thread vram_logger([device_id]() {
        while (true) {
            log_vram_usage(device_id);
            std::this_thread::sleep_for(std::chrono::seconds(30));
        }
    });
    vram_logger.detach();

    uint32_t* d_result;
    err = hipMalloc(&d_result, sizeof(uint32_t));
    if (err != hipSuccess) {
        cuda_log("ERROR", "31", "Failed to allocate device memory: " + 
                               std::string(hipGetErrorString(err)));
        hipFree(d_dag);
        ethash_light_delete(light);
        return;
    }
    cuda_log("CUDA", "36", "Allocated device memory for results");

    err = hipMemset(d_result, 0, sizeof(uint32_t));
    if (err != hipSuccess) {
        cuda_log("ERROR", "31", "Failed to initialize device memory: " + 
                               std::string(hipGetErrorString(err)));
        hipFree(d_result);
        hipFree(d_dag);
        ethash_light_delete(light);
        return;
    }

    dim3 threads_per_block(256);
    dim3 num_blocks(intensity * 1024);
    cuda_log("CUDA", "36", "Launching kernel with " + std::to_string(num_blocks.x) + 
                          " blocks, " + std::to_string(threads_per_block.x) + " threads per block");

    kawpow_kernel<<<num_blocks, threads_per_block>>>(header_hash, start_nonce, d_result, d_dag, dag_size);
    
    err = hipGetLastError();
    if (err != hipSuccess) {
        cuda_log("ERROR", "31", "Kernel launch failed: " + 
                               std::string(hipGetErrorString(err)));
        hipFree(d_result);
        hipFree(d_dag);
        ethash_light_delete(light);
        return;
    }
    cuda_log("CUDA", "36", "Kernel launched successfully");

    uint32_t h_result = 0;
    err = hipMemcpy(&h_result, d_result, sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        cuda_log("ERROR", "31", "Failed to copy result from device: " + 
                               std::string(hipGetErrorString(err)));
        hipFree(d_result);
        hipFree(d_dag);
        ethash_light_delete(light);
        return;
    }

    if (h_result != 0) {
        cuda_log("CUDA", "36", "Found valid nonce: " + std::to_string(h_result));
        // In a real miner, you would submit this nonce to the stratum server.
    }

    err = hipFree(d_result);
    if (err != hipSuccess) {
        cuda_log("ERROR", "31", "Failed to free device memory: " + 
                               std::string(hipGetErrorString(err)));
        hipFree(d_dag);
        ethash_light_delete(light);
        return;
    }
    hipFree(d_dag);
    ethash_light_delete(light);
    cuda_log("CUDA", "36", "Search completed on device " + std::to_string(device_id));
}

